#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//                                                                            //
//   Created: 2019-01-25                                                      //
//    Author: Marcos Romero Lamas (mromerol@cern.ch)                          //
//                                                                            //
//    This file is part of phis-scq packages, Santiago's framework for the    //
//                     phi_s analysis in Bs -> Jpsi K+ K-                     //
//                                                                            //
////////////////////////////////////////////////////////////////////////////////


#include <lib99ocl/core.h>
#include <lib99ocl/random.h>

#include "time_angular_distribution.h"
#include "decay_time_acceptance.h"
#include "angular_acceptance.h"
#include "cross_rate_bs.h"


// Generate toy {{{

KERNEL
void dG5toy(/* args {{{ */GLOBAL_MEM ftype * out,
    const ftype G, const ftype DG, const ftype DM,
    GLOBAL_MEM const ftype * CSP,
    //GLOBAL_MEM const ftype * CSD,
    //GLOBAL_MEM const ftype * CPD,
    GLOBAL_MEM const ftype * ASlon,
    GLOBAL_MEM const ftype * APlon,
    GLOBAL_MEM const ftype * APpar,
    GLOBAL_MEM const ftype * APper,
    //const ftype ADlon, const ftype ADpar, const ftype ADper,
    const ftype pSlon,
    const ftype pPlon, const ftype pPpar, const ftype pPper,
    //const ftype pDlon, const ftype pDpar, const ftype pDper,
    GLOBAL_MEM const ftype *dSlon,
    const ftype dPlon, const ftype dPpar, const ftype dPper,
    //const ftype dDlon, const ftype dDpar, const ftype dDper,
    const ftype lSlon,
    const ftype lPlon, const ftype lPpar, const ftype lPper,
    //const ftype lDlon, const ftype lDpar, const ftype lDper,
    // Time limits
    const ftype tLL, const ftype tUL,
    const ftype cosKLL, const ftype cosKUL,
    const ftype cosLLL, const ftype cosLUL,
    const ftype hphiLL, const ftype hphiUL,
    // Time resolution
    const ftype sigma_offset, const ftype sigma_slope, const ftype sigma_curvature,
    const ftype mu,
    // Flavor tagging
    const ftype eta_bar_os, const ftype eta_bar_ss,
    const ftype p0_os,  const ftype p1_os, const ftype p2_os,
    const ftype p0_ss,  const ftype p1_ss, const ftype p2_ss,
    const ftype dp0_os, const ftype dp1_os, const ftype dp2_os,
    const ftype dp0_ss, const ftype dp1_ss, const ftype dp2_ss,
    // Time acceptance
    GLOBAL_MEM const ftype *coeffs,
    // Angular acceptance
    GLOBAL_MEM const ftype *tijk,
    const int order_cosK, const int order_cosL, const int order_hphi,
    const int USE_FK, const int BINS, const int USE_ANGACC, const int USE_TIMEACC,
    const int USE_TIMEOFFSET, const int SET_TAGGING, const int USE_TIMERES,
    const ftype PROB_MAX, const int SEED, const int NEVT /*}}}*/)
{
  int evt = get_global_id(0);
  if (evt >= NEVT) { return; }

  // auxiliar variables
  ftype threshold = 0.0;
  ftype iter = 0.0;

  // Prepare random number generation {{{

  #ifdef CUDA
    hiprandState state;
    hiprand_init((unsigned long long)clock(), evt, 0, &state);
  #else
    int _seed = SEED;
    int *state = &_seed;
  #endif

  // }}}


  // Decay time resolution {{{

  ftype sigmat = 0.0;
  if (USE_TIMERES)
  {
    sigmat = rng_log_normal(-3.22, 0.309, &state);
  }

  // }}}


  // Flavor tagging {{{

  ftype qOS = 0;
  ftype qSS = 0;
  ftype etaOS = 0.5;
  ftype etaSS = 0.5;

  if (SET_TAGGING == 1) // DATA
  {
    // generate qOS
    ftype tag = rng_uniform(&state);
    if      (tag < 0.175) { qOS =  1.; }
    else if (tag < 0.350) { qOS = -1.; }
    else                  { qOS =  0.; }
    // generate qSS
    tag = rng_uniform(&state);
    if      (tag < 0.330) { qSS =  1.; }
    else if (tag < 0.660) { qSS = -1.; }
    else                  { qSS =  0.; }

    ftype OSmax = tagOSgen(0.5);
    ftype SSmax = tagSSgen(0.5);

    // generate etaOS
    if (qOS > 0.5 || qOS < -0.5)
    {
      while(1)
      {
        tag = 0.49*rng_uniform(&state);
        threshold = OSmax*rng_uniform(&state);
        if (tagOSgen(tag) > threshold) break;
      }
      etaOS = tag;
    }
    // generate etaSS
    if (qSS > 0.5 || qSS < -0.5)
    {
      while(1)
      {
        tag = 0.49*rng_uniform(&state);
        threshold = SSmax*rng_uniform(&state);
        if (tagSSgen(tag) > threshold) break;
      }
      etaSS = tag;
    }
  }
  else if (SET_TAGGING == 0) // PERFECT, MC
  {
    ftype tag = rng_uniform(&state);
    if (tag < 0.5) {qOS = +1.0; qSS = +1.0;}
    else           {qOS = -1.0; qSS = -1.0;}
    etaOS = 0.5; etaSS = 0.5;
  }
  else //TRUE
  {
    qOS = 0.0; qSS = 0.0;
    etaOS = 0.5; etaSS = 0.5;
  }
  qOS *= 531; // put same number used in EvtGen 
  qSS *= 531; // put same number used in EvtGen

  // }}}


  // Loop and generate {{{

  ftype cosK, cosL, hphi, time, pdf, angacc;
  const ftype mass = out[evt*10+4];
  const unsigned int bin = BINS>1 ? getMassBin(mass) : 0;

  while(1)
  {
    // Random numbers
    cosK = - 1.0  +    2.0*rng_uniform(&state);
    cosL = - 1.0  +    2.0*rng_uniform(&state);
    hphi = - M_PI + 2*M_PI*rng_uniform(&state);
    time =   tLL  -    log(rng_uniform(&state)) / (G-0.5*DG);

    //WARN!
    //cosK=+0.45774651;
    //cosL=-0.20873141;
    //hphi=-0.00354153;
    //time=+3.2032610199999998;
    //sigmat=0.04120890;
      /* printf("cosK=%lf cosL=%lf hphi=%lf time=%lf sigmat=%lf qOS=%lf qSS=%lf etaOS=%lf etaSS=%lf", cosK, cosL, hphi, time, sigmat, qOS, qSS, etaOS, etaSS); */

    // PDF threshold
    threshold = PROB_MAX * rng_uniform(&state);

    // Prepare data and pdf variables to DiffRate CUDA function
    ftype data[9] = {cosK, cosL, hphi, time, sigmat, qOS, qSS, etaOS, etaSS};
    pdf = 0.0;

    // Get pdf value from angular distribution
    // if time is larger than asked, put pdf to zero
    if ((time >= tLL) & (time <= tUL))
    {
      // Time acceptance is already included in the nominal p.d.f. Thus we do
      // not need to precompte it here, and it is handled by USE_TIMEACC as
      // usually. However, angular accepntace needs to be precompute, since
      // it is not present in the p.d.f. and we need to include it. Angular
      // acceptance is handled with USE_ANGACC as usually too.
      angacc = 1.0;
      if (USE_ANGACC)
      {
        //angacc = ang_eff(data[0], data[1], data[2], tijk);
        //if (evt  == 0){ printf("angacc = %f    ", angacc);}
        angacc = angular_efficiency_weights(data[0], data[1], data[2], tijk);
        //if (evt  == 0){ printf("angacc = %f\n ", angacc);}
      }

      pdf = rateBs(data,
          G, DG, DM, CSP[bin],
          ASlon[bin], APlon[bin], APpar[bin], APper[bin],
          pSlon,      pPlon,      pPpar,      pPper,
          dSlon[bin], dPlon,      dPpar,      dPper,
          lSlon,      lPlon,      lPpar,      lPper,
          tLL, tUL, cosKLL, cosKUL, cosLLL, cosLUL, hphiLL, hphiUL,
          sigma_offset, sigma_slope, sigma_curvature, mu,
          eta_bar_os, eta_bar_ss,
          p0_os,  p1_os, p2_os,
          p0_ss,  p1_ss, p2_ss,
          dp0_os, dp1_os, dp2_os,
          dp0_ss, dp1_ss, dp2_ss,
          coeffs,
          tijk,
          USE_FK, USE_ANGACC, USE_TIMEACC,
          USE_TIMEOFFSET, SET_TAGGING, USE_TIMERES);
      pdf *= angacc * exp((G-0.5*DG)*(time-tLL));
      pdf *= (G-0.5*DG) * (1 - exp((G-0.5*DG)*(-tUL+tLL)));
    }

    // }}}


    // final checks {{{

    // check if probability is greater than the PROB_MAX
    if ( (pdf > PROB_MAX) && (get_global_id(0)<100) ) 
    {
      printf("WARNING: PDF [=%f] > PROB_MAX [=%f]\n", pdf, PROB_MAX);
    }

    // stop if it's taking too much iterations 
    iter++;
    if( (iter > 100000) && (get_global_id(0)<100) )
    {
      printf("ERROR: This p.d.f. is too hard...");
      return;
    }

    // }}}


    // Store generated and accepted values in array {{{

    if (pdf >= threshold)
    {
      out[evt*10+0] = data[0]; // cosK
      out[evt*10+1] = data[1]; // cosL
      out[evt*10+2] = data[2]; // hphi
      out[evt*10+3] = data[3]; // time
      // mass (index 4) is already in the array :)
      out[evt*10+5] = data[4]; // sigma_t
      out[evt*10+6] = data[5]; // qOS
      out[evt*10+7] = data[6]; // qSS
      out[evt*10+8] = data[7]; // etaOS
      out[evt*10+9] = data[8]; // etaSS
      return;
    }

    // }}}

  }

}

// }}}


// vim: fdm=marker 
